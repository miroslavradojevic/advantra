
#include <hip/hip_runtime.h>
extern "C"
__global__ void createKernels(
float* kernels,
int size,
int nrOfOrientations,
int nrOfScales,
float sigma_min,
int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int orientation = threadIdx.x;
    int scale = blockIdx.x;

    if (index < N)
    {
        int s2 = size / 2;
        int nn = 11;
        float gamma = 20;
        // see: http://en.wikipedia.org/wiki/Gabor_filter

        float alpha = (3.141592654 * orientation) / nrOfOrientations;
        float sigma_adjusted = 0.6 * sigma_min * scale + sigma_min;
        float s2d_min = 2 * sigma_adjusted * sigma_adjusted;
        float s2d_max = 200;

        float lambda = 5 * sigma_adjusted;

        float totalSum = 0;
        for (int j = -s2; j <= s2; j++) {
            for (int i = -s2; i <= s2; i++) {
                float sum = 0;
                for (int ii = 0; ii < nn; ii++) {
                    float xx = i - 0.5 + (1 + 2 * ii) / (2.0 * nn);
                    for (int jj = 0; jj < nn; jj++)
                    {
                        float yy = j - 0.5 + (1 + 2 * jj) / (2.0 * nn);

                        float xx_ = yy * sinf(alpha) + xx * cosf(alpha);
                        float yy_ = yy * cosf(alpha) - xx * sinf(alpha);

                        //sum += expf(-(xx_ * xx_) / s2d_max - (yy_ * yy_) / s2d_min) * cosf(2 * 3.141592654 * yy_ / (size));
                        // Gabor filter
                        sum += expf(-(xx_ * xx_) / s2d_min / gamma  - (yy_ * yy_) / s2d_min) * cosf(2 * 3.141592654 * yy_ / lambda);
                    }
                }

                kernels[(i + s2) + (j + s2) * size + size * size * orientation + size * size * nrOfOrientations * scale] = sum;
                totalSum += sum;
            }
        }
        for (int j = -s2; j <= s2; j++) {
            for (int i = -s2; i <= s2; i++) {
                //kernels[(i + s2) + (j + s2) * size + size * size * orientation + size * size * nrOfOrientations * scale] /= totalSum;
            }
        }
    }
}

extern "C"
__global__ void applyKernels(
float* kernels,
float* inimg,
float* outimg,
int* positions,
int size,
int nrOfOrientations,
int scale,
int dimsx,
int dimsy,
int N_threads,
int NN)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = index;

    while (tid < NN)
    {
        int x = positions[2 * tid];
        int y = positions[2 * tid + 1];
        int s2 = size / 2;

        int bestOrientation = 0;
        float bestResponse = 0;
        for (int r = 0; r < nrOfOrientations; r++) {
            float sum = 0;
            for (int j = -s2; j <= s2; j++) {
                for (int i = -s2; i <= s2; i++) {
                    sum += inimg[x + i + dimsx * (y + j)] * kernels[i + s2 + size * (j + s2) + size * size * r + size * size * nrOfOrientations * scale];
                }
            }
            if (sum > bestResponse) {
                bestResponse = sum;
                bestOrientation = r;
            }
        }
        if (outimg[x + dimsx * y] < bestResponse)
        {
            outimg[x + dimsx * y] = bestResponse;
        }

        tid += N_threads;
    }

 }