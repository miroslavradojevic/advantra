
#include <hip/hip_runtime.h>
extern "C"
__global__ void applyKernels(   float* kernels_hat,
                                float* kernels_hat_sum_2,
                                float* inimg,
                                float* pos2zncc,
                                float* pos2sigma,
                                float* pos2vx,
                                float* pos2vy,
                                int*   pos,
                                int L,
                                int N,
                                int scale,
                                int dimsx,
                                int dimsy,
                                int N_threads,
                                int NN)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = index;

    while (tid < NN)
    {

        int x = pos[2 * tid];
        int y = pos[2 * tid + 1];
        int L2 = L / 2;

        float imgVal;
        float imgValsAvg = 0;

        for (int pidx = 0; pidx < L*L; pidx++) {

            int xx = x + (pidx % L) - L2;
            int yy = y + (pidx / L) - L2;

            imgVal = (xx>=0 && xx<dimsx && yy>=0 && yy<dimsy)?inimg[xx + yy * dimsx]:0;

            imgValsAvg += imgVal;

        }

        imgValsAvg /= (float) (L*L);

        for (int didx = 0; didx < N; didx++) {

            float num = 0;
            float den = 0;
            float zncc;

            for (int pidx = 0; pidx < L*L; pidx++) {

                // sample from the image
                int xx = x + (pidx % L) - L2;
                int yy = y + (pidx / L) - L2;

                imgVal = (xx>=0 && xx<dimsx && yy>=0 && yy<dimsy)?inimg[xx + yy * dimsx]:0;
 
                num += (imgVal - imgValsAvg) * kernels_hat[scale*N*L*L + didx*L*L + pidx];
                den += (imgVal - imgValsAvg) * (imgVal - imgValsAvg);

            }

            zncc = num / (float) sqrtf(den * kernels_hat_sum_2[didx + scale*N]);

            if (zncc > pos2zncc[tid]) {
                
                pos2zncc[tid] = zncc;
                
                pos2sigma[tid] = scale;

                float ang = didx * (3.141592654 / N);

                float vx = -sinf(ang);
                float vy =  cosf(ang);

                pos2vx[tid] = vx;
                pos2vy[tid] = vy; 

            }

        }

        tid += N_threads;

    }

 }